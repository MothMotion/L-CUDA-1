#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(arr_t* arr, arrO_t& out, const uint32_t& size) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t* d_arr;
  arrO_t* d_out;
  uint32_t* d_size;
  hipMalloc((void**)&d_arr, size*sizeof(arr_t));
  hipMalloc((void**)&d_out, sizeof(arrO_t));
  hipMalloc((void**)&d_size, sizeof(uint32_t));

  hipStream_t stream;
  hipStreamCreate(&stream);


  CUDATIME(({
    hipHostRegister(arr, size, hipHostRegisterDefault);

    hipMemcpyAsync(d_arr, arr, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemset(d_out, 0, sizeof(arrO_t));
    hipMemcpy(d_size, &size, sizeof(arr_t), hipMemcpyHostToDevice);

    hipHostUnregister(arr);
  }), time.memcpy, start, end);

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  CUDATIME(({
    KSum<<<blocks, threads>>>(d_arr, size, *d_out);
  }), time.run, start, end);

  CUDATIME(({ 
    hipMemcpy(&out, &d_out, sizeof(arrO_t), hipMemcpyDeviceToHost);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr); 
  hipFree(d_out);
  hipFree(d_size);

  return time;
}



__global__ void KSum(arr_t* arr, const uint32_t& size, arrO_t* out) {
  const uint32_t proc_size = size / gridDim.x;
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x * proc_size;

  for(uint32_t i=idx; i<idx+proc_size && i<size; ++i)
    out[threadIdx.x] += arr[i];
}

#endif
