#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(arr_t* arr, arrO_t& out, const uint32_t& size) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t* d_arr;
  arrO_t* d_out;
  // {size, options}
  uint32_t *d_size, *d_counter;
  hipMalloc((void**)&d_arr, size*sizeof(arr_t));
  hipMalloc((void**)&d_out, sizeof(arrO_t));
  hipMalloc((void**)&d_size, sizeof(uint32_t));
  hipMalloc((void**)&d_counter, sizeof(uint32_t));

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  hipStream_t stream;
  hipStreamCreate(&stream);


  CUDATIME(({
    hipHostRegister(arr, size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(d_arr, arr, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemset(d_out, 0, sizeof(arrO_t));
    hipMemset(d_counter, 0, sizeof(uint32_t));
    hipMemcpy(d_size, &size, sizeof(uint32_t), hipMemcpyHostToDevice); 

    hipHostUnregister(arr);
  }), time.memcpy, start, end); 

  CUDATIME(({
    KSum<<<blocks, threads>>>(d_arr, *d_size, *d_out, *d_counter); 
  }), time.run, start, end);

  CUDATIME(({ 
    hipMemcpy(&out, d_out, sizeof(arrO_t), hipMemcpyDeviceToHost);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr); 
  hipFree(d_out);
  hipFree(d_size);
  hipFree(d_counter);

  return time;
}



__global__ void KSum(arr_t* arr, const uint32_t& size, arrO_t& out, uint32_t& counter) {
  const uint32_t proc_size = size / gridDim.x;
  const uint32_t thread_size = proc_size / blockDim.x;
  uint32_t idx = blockIdx.x * proc_size + threadIdx.x * thread_size;

  arrO_t temp_out = 0;

  for(uint32_t i=idx; i<idx+thread_size && i<size; ++i)
    temp_out += arr[i];

  __shared__ arrO_t sdata[MAX_THREADS];
  sdata[threadIdx.x] = temp_out;

  __syncthreads();

  for(uint32_t i = blockDim.x/2; i>0; i >>= 1) {
    if(threadIdx.x < i)
      sdata[threadIdx.x] += sdata[threadIdx.x + i];
    __syncthreads();
  }

  if(threadIdx.x == 0) {
    while(atomicInc(&counter, gridDim.x) != blockIdx.x)
      __threadfence();

    out += sdata[0];
    __threadfence();
  }
}

#endif
