#include "hip/hip_runtime.h"
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(arr_t* arr, arrO_t& out, const uint32_t& size) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  struct {uint32_t size; uint32_t threads;}
  options = {size, KTHREADS};

  arr_t* d_arr;
  arrO_t* d_out;
  // {size, options}
  uint32_t *d_options;
  hipMalloc((void**)&d_arr, size*sizeof(arr_t));
  hipMalloc((void**)&d_out, KTHREADS*sizeof(arrO_t));
  hipMalloc((void**)&d_options, 2*sizeof(uint32_t));

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  hipStream_t stream;
  hipStreamCreate(&stream);


  CUDATIME(({
    hipHostRegister(arr, size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(d_arr, arr, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemset(d_out, 0, KTHREADS*sizeof(arrO_t));
    hipMemcpy(d_options, &options, 2*sizeof(uint32_t), hipMemcpyHostToDevice); 

    hipHostUnregister(arr);
  }), time.memcpy, start, end); 

  CUDATIME(({
    KSum<<<blocks, threads>>>(d_arr, d_options[0], d_out); 
  }), time.run, start, end);

  CUDATIME(({ 
    hipMemcpy(&out, d_out, sizeof(arrO_t), hipMemcpyDeviceToHost);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr); 
  hipFree(d_out);
  hipFree(d_options);

  return time;
}



__global__ void KSum(arr_t* arr, const uint32_t& size, arrO_t* out) {
  const uint32_t proc_size = size / gridDim.x;
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x * proc_size;

  for(uint32_t i=idx; i<idx+proc_size && i<size; ++i)
    out[threadIdx.x] += arr[i];

  __syncthreads();

  if(blockIdx.x == gridDim.x - 1 && threadIdx.x == 0)
    for(uint32_t i=1; i<blockDim.x; ++i)
      out[0] += out[i];
}

#endif
