#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#ifndef SERIAL



#include "config.h"
#include "timer.h"
#include "vec_oper.h"
#include "k_vec_oper.h"

#include <stdint.h>
#include <stddef.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



time_s Operation(arr_t* arr, arrO_t& out, const uint32_t& size) {
  time_s time;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  arr_t* d_arr;
  arrO_t* d_out;
  // {size, options}
  uint32_t *d_size, *d_counter;
  hipMalloc((void**)&d_arr, size*sizeof(arr_t));
  hipMalloc((void**)&d_out, sizeof(arrO_t));
  hipMalloc((void**)&d_size, sizeof(uint32_t));
  hipMalloc((void**)&d_counter, sizeof(uint32_t));

  dim3 blocks(KBLOCKS, 1, 1);
  dim3 threads(KTHREADS, 1, 1);

  hipStream_t stream;
  hipStreamCreate(&stream);


  CUDATIME(({
    hipHostRegister(arr, size*sizeof(arr_t), hipHostRegisterDefault);

    hipMemcpyAsync(d_arr, arr, size*sizeof(arr_t), hipMemcpyHostToDevice, stream);
    hipMemset(d_out, 0, blocks.x*sizeof(arrO_t));
    hipMemcpy(d_size, &size, sizeof(uint32_t), hipMemcpyHostToDevice); 

    hipHostUnregister(arr);
  }), time.memcpy, start, end); 

  CUDATIME(({
    while(blocks.x > threads.x * threads.x) {
      if(blocks.x == KBLOCKS)
        KSum<<<blocks, threads>>>(d_arr, *d_size, d_out);
      else
        KSum<<<blocks, threads>>>((arrO_t*)d_arr, *d_size, d_out);

      hipMemcpy(d_arr, d_out, blocks.x * sizeof(arrO_t), hipMemcpyDeviceToDevice);
      blocks.x = (blocks.x + threads.x - 1)/threads.x;
      hipMemset(d_out, 0, blocks.x * sizeof(arrO_t)); 
    }
    KSum<<<1, threads>>>((arrO_t*)d_arr, *d_size, d_out);
  }), time.run, start, end);

  CUDATIME(({ 
    hipMemcpy(&out, d_out, sizeof(arrO_t), hipMemcpyDeviceToHost);
  }), time.memret, start, end); 

  time.total = time.memcpy + time.run + time.memret;

  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipFree(d_arr); 
  hipFree(d_out);
  hipFree(d_size);
  hipFree(d_counter);

  return time;
}



template<typename T, typename M>
__global__ void KSum(T* arr, const uint32_t& size, M* out) {
  const uint32_t proc_size = size / gridDim.x;
  const uint32_t thread_size = proc_size / blockDim.x;
  uint32_t idx = blockIdx.x * proc_size + threadIdx.x * thread_size;

  arrO_t temp_out = 0;

  for(uint32_t i=idx; i<idx+thread_size && i<size; ++i)
    temp_out += arr[i];

  __shared__ arrO_t sdata[MAX_THREADS];
  sdata[threadIdx.x] = temp_out;

  __syncthreads();

  for(uint32_t i = blockDim.x/2; i>0; i >>= 1) {
    if(threadIdx.x < i)
      sdata[threadIdx.x] += sdata[threadIdx.x + i];
    __syncthreads();
  }

  if(threadIdx.x == 0)
    out[blockIdx.x] += sdata[0];
}

#endif
